#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <chrono>
#include <stdio.h>
#include <ole2.h>
#include <olectl.h>
#include <string>
//#include <atlbase.h>
#include <WinUser.h>
#include <iostream>

#define GLOBAL_SIZE 2

#define CAPTURE_WIDTH 871
#define CAPTURE_HEIGHT 747
#define CAPTURE_X_OFFSET 10
#define CAPTURE_Y_OFFSET 40

#define LINE_Y_OFFSET -50.1
#define LINE_X_OFFSET 2

#define TOTAL_THREADS 1

int iterClicked = 0;
int numberOfThreads = 0;
bool threadsInUse = false;
bool saveBitmap(LPCSTR filename, HBITMAP bmp, HPALETTE pal);
void mouseClick(int px, int py);

class pixel {
public:
    unsigned char r, g, b;
    pixel(unsigned char r, unsigned char g, unsigned char b) {
        this->r = r; this->g = g; this->b = b;
    }
    pixel() {}
    std::string disp() {
        return "(" + std::to_string(int(r)) + "," + std::to_string(int(g)) + "," + std::to_string(int(b)) + ")";
    }
};
pixel** buffer;
pixel* buffer1D;
int bufferHeight=0, bufferWidth=0;

__global__ void scanAndClickKernel(pixel* grid, int gridHeight, int gridWidth, int* sol) {
    int offsetX = LINE_X_OFFSET;
    float offsetY = LINE_Y_OFFSET;
    for (int i = 0; i < gridHeight; i++) {
        for (int j = 0; j < gridWidth; j++) {
            if (grid[i * gridWidth + j].r == 255 && grid[i * gridWidth + j].g == 0 && grid[i * gridWidth + j].b == 0) {
                //++iterClicked;
                //if (iterClicked == 1) {
                //mouseClick(int(float(j + offsetX + CAPTURE_X_OFFSET) / 1920 * 65535), int(float(gridHeight - (i + offsetY + CAPTURE_Y_OFFSET)) / 1080 * 65535));
                //}
                sol[0] = j; sol[1] = i;
                return;
            }
        }
    }
}

hipError_t targetWithCuda(pixel* grid, unsigned int height, unsigned int width, int* sol);

unsigned char* readBitmapFile(std::string filename, int padding = 1) {
    FILE* f;
    fopen_s(&f, filename.c_str(), "rb");
    unsigned char info[54];

    // read the 54-byte header
    fread(info, sizeof(unsigned char), 54, f);

    // extract image height and width from header
    int width = *(int*)&info[18];
    int height = *(int*)&info[22];
    bufferHeight = width; bufferHeight = height;
    buffer = new pixel*[height];
    for (int i = 0; i < height;i++) {
        buffer[i] = new pixel[width];
    }// std::vector<std::vector<pixel>>(height, std::vector<pixel>(width));

    // read the rest of the data at once
    unsigned char* data = new unsigned char[width * 3 + padding];
    for (int i = 0; i < height; i++) {
        fread(data, sizeof(unsigned char), width * 3 + padding, f);
        for (int j = 0; j < width * 3; j += 3) {
            buffer[i][j / 3] = pixel(data[j + 2], data[j + 1], data[j]);
        }
    }
    fclose(f);

    return data;
}

void readBitmapFileAs1DArray(std::string filename, int padding = 1) {
    FILE* f;
    fopen_s(&f, filename.c_str(), "rb");
    unsigned char info[54];

    // read the 54-byte header
    fread(info, sizeof(unsigned char), 54, f);

    // extract image height and width from header
    int width = *(int*)&info[18];
    int height = *(int*)&info[22];
    bufferWidth = width; bufferHeight = height;
    buffer1D = new pixel[height*width];
    int size = 3 * width * height;

    // read the rest of the data at once
    unsigned char* data = new unsigned char[width * 3 + padding];
    for (int i = 0; i < height; i++) {
        fread(data, sizeof(unsigned char), width * 3 + padding, f);
        for (int j = 0; j < width * 3; j += 3) {
            buffer1D[i*width + j / 3] = pixel(data[j + 2], data[j + 1], data[j]);
        }
    }
    fclose(f);

    return;
}

bool screenCapturePart(int x, int y, int w, int h, LPCSTR fname) {
    HDC hdcSource = GetDC(NULL);
    HDC hdcMemory = CreateCompatibleDC(hdcSource);

    int capX = GetDeviceCaps(hdcSource, HORZRES);
    int capY = GetDeviceCaps(hdcSource, VERTRES);

    HBITMAP hBitmap = CreateCompatibleBitmap(hdcSource, w, h);
    HBITMAP hBitmapOld = (HBITMAP)SelectObject(hdcMemory, hBitmap);

    BitBlt(hdcMemory, 0, 0, w, h, hdcSource, x, y, SRCCOPY);
    hBitmap = (HBITMAP)SelectObject(hdcMemory, hBitmapOld);

    DeleteDC(hdcSource);
    DeleteDC(hdcMemory);

    HPALETTE hpal = NULL;
    if (saveBitmap(fname, hBitmap, hpal)) return true;
    return false;
}

bool saveBitmap(LPCSTR filename, HBITMAP bmp, HPALETTE pal) {
    bool result = false;
    PICTDESC pd;

    pd.cbSizeofstruct = sizeof(PICTDESC);
    pd.picType = PICTYPE_BITMAP;
    pd.bmp.hbitmap = bmp;
    pd.bmp.hpal = pal;

    LPPICTURE picture;
    HRESULT res = OleCreatePictureIndirect(&pd, IID_IPicture, false,
        reinterpret_cast<void**>(&picture));

    if (!SUCCEEDED(res))
        return false;

    LPSTREAM stream;
    res = CreateStreamOnHGlobal(0, true, &stream);

    if (!SUCCEEDED(res)) {
        picture->Release();
        return false;
    }


    LONG bytes_streamed;
    res = picture->SaveAsFile(stream, true, &bytes_streamed);

    //USES_CONVERSION;
    HANDLE file = CreateFile((filename), GENERIC_WRITE, FILE_SHARE_READ, 0,
        CREATE_ALWAYS, FILE_ATTRIBUTE_NORMAL, 0);

    if (!SUCCEEDED(res) || !file) {
        stream->Release();
        picture->Release();
        return false;
    }

    HGLOBAL mem = 0;
    GetHGlobalFromStream(stream, &mem);
    LPVOID data = GlobalLock(mem);

    DWORD bytes_written;

    result = !!WriteFile(file, data, bytes_streamed, &bytes_written, 0);
    result &= (bytes_written == static_cast<DWORD>(bytes_streamed));

    GlobalUnlock(mem);
    CloseHandle(file);

    stream->Release();
    picture->Release();

    return result;
}

void mouseClick(int px, int py) {
    INPUT Inputs[3] = { 0 };

    Inputs[0].type = INPUT_MOUSE;
    Inputs[0].mi.dwFlags = MOUSEEVENTF_ABSOLUTE | MOUSEEVENTF_MOVE | MOUSEEVENTF_MOVE_NOCOALESCE;
    Inputs[0].mi.dx = px; // desired X coordinate
    Inputs[0].mi.dy = py; // desired Y coordinate

    Inputs[1].type = INPUT_MOUSE;
    Inputs[1].mi.dwFlags = MOUSEEVENTF_LEFTDOWN;

    Inputs[2].type = INPUT_MOUSE;
    Inputs[2].mi.dwFlags = MOUSEEVENTF_LEFTUP;

    std::cout << "click at " << "(" << px << ", " << py << ")" << std::endl;
    SendInput(3, Inputs, sizeof(INPUT));
}

void threadFunction(int id, int len, int totalRows, int totalThreads, int iter) {
    int sharestart = id * totalRows / totalThreads;
    int shareend = min(id * totalRows / totalThreads + (totalRows / totalThreads) - 1, totalRows - 1);
    for (int i = sharestart; i <= shareend; i++) {
        if (iterClicked > 0)return;
        for (int j = 0; j < len; j++) {
            if (iterClicked > 0)return;
            if (buffer[i][j].r == 255 && buffer[i][j].g == 0 && buffer[i][j].b == 0) {
                ++iterClicked;
                if (iterClicked == 1) {
                    //mouseClick(int(float(j + offsetX + CAPTURE_X_OFFSET) / 1920 * 65535), int(float(totalRows - (i + offsetY + CAPTURE_Y_OFFSET)) / 1080 * 65535));
                    std::cout << j <<"-" << i << std::endl;
                }
                return;
            }
        }
    }
}

int main() {
    /*
    const int arraySize = GLOBAL_SIZE;
    int a[arraySize] = { 1, 2 };// , 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5};
    int b[arraySize] = { 10, 20 };// , 30, 40, 50, 10, 20, 30, 40, 50, 10, 20, 30, 40, 50, 10, 20, 30, 40, 50, 10, 20, 30, 40, 50, 10, 20, 30, 40, 50, 10, 20, 30, 40, 50, 10, 20, 30, 40, 50};
    int c[arraySize] = { 0 };
    int d[arraySize] = { 0 };

    auto start = std::chrono::high_resolution_clock::now();
    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    auto end = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    std::cout << "time taken by GPU : " << duration.count() << std::endl;

    start = std::chrono::high_resolution_clock::now();
    addCPU(d, a, b, arraySize);
    end = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    std::cout << "time taken by CPU : " << duration.count() << std::endl;

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    */

    mouseClick(5000, 700);
    Sleep(1000);
    //mouseClick(int(float(10)/1920*65535), int(float(39) / 1080 * 65535));
    //Sleep(20);
    //mouseClick(int(float(871) / 1920 * 65535), int(float(787) / 1080 * 65535));

    int offsetX = LINE_X_OFFSET;
    float offsetY = LINE_Y_OFFSET;

    std::string filename = "screenshots/multithreadcup";
    //std::thread threadRow[TOTAL_THREADS];
    for (int iter = 1; iter <= 11; iter++) {
        screenCapturePart(CAPTURE_X_OFFSET, CAPTURE_Y_OFFSET, CAPTURE_WIDTH, CAPTURE_HEIGHT, (filename + std::to_string(iter) + ".bmp").c_str());
        readBitmapFileAs1DArray((filename + std::to_string(iter) + ".bmp"), 3);
        int* sol=new int[2];
        
        hipError_t cudaStatus = targetWithCuda(buffer1D, bufferHeight, bufferWidth, sol);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addWithCuda failed!");
            return 1;
        }

        mouseClick(int(float(sol[0] + offsetX + CAPTURE_X_OFFSET) / 1920 * 65535), int(float(bufferHeight - (sol[1] + offsetY + CAPTURE_Y_OFFSET)) / 1080 * 65535));

        //avgX /= avgCounter; avgY /= avgCounter;
        //avgY = buffer.size() - avgY;
        //std::cout << avgX << " " << avgY << std::endl;
        //mouseClick(int(float(avgX + 10) / 1920 * 65535), int(float(avgY + 40) / 1080 * 65535));
    }

    return 0;
}

hipError_t targetWithCuda(pixel* grid, unsigned int height, unsigned int width, int *sol) {
    int* cudaSol = 0;
    pixel* cudaGrid = 0;
    hipError_t cudaStatus;
    unsigned int size = 1;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&cudaGrid, height * width * sizeof(pixel));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&cudaSol, 2 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(cudaGrid, grid, height * width * sizeof(pixel), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(cudaSol, sol, 2 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    scanAndClickKernel <<<1, size >>> (cudaGrid, height, width, cudaSol);
    //auto end = std::chrono::high_resolution_clock::now();
    //auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    //std::cout << "time taken by GPU : " << duration.count() << std::endl;

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    
    cudaStatus = hipMemcpy(sol, cudaSol, 2 * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    

    Error:
    hipFree(cudaGrid);

    return cudaStatus;
}
